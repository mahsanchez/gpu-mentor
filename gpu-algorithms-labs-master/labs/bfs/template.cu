#include "hip/hip_runtime.h"
#include <cstdio>
#include <cstdlib>
#include <stdio.h>

#include "template.hu"

#define BLOCK_SIZE 512
#define WARP_SIZE 32
#define NUM_WARPS (BLOCK_SIZE / WARP_SIZE)

// Maximum number of elements that can be inserted into a block queue
#define BQ_CAPACITY 2048

// Maximum number of elements that can be inserted into a warp queue
#define WQ_CAPACITY 128

/******************************************************************************
 GPU kernels
*******************************************************************************/

__global__ void gpu_global_queuing_kernel(unsigned int *nodePtrs,
                                          unsigned int *nodeNeighbors,
                                          unsigned int *nodeVisited,
                                          unsigned int *currLevelNodes,
                                          unsigned int *nextLevelNodes,
                                          unsigned int *numCurrLevelNodes,
                                          unsigned int *numNextLevelNodes) {

  // INSERT KERNEL CODE HERE
  // Loop over all nodes in the curent level
  // Loop over all neighbors of the node
  // If the neighbor hasn't been visited yet
  // Add it to the global queue
}

__global__ void gpu_block_queuing_kernel(unsigned int *nodePtrs,
                                         unsigned int *nodeNeighbors,
                                         unsigned int *nodeVisited,
                                         unsigned int *currLevelNodes,
                                         unsigned int *nextLevelNodes,
                                         unsigned int *numCurrLevelNodes,
                                         unsigned int *numNextLevelNodes) {
  // INSERT KERNEL CODE HERE

  // Initialize shared memory queue

  // Loop over all nodes in the curent level
  // Loop over all neighbors of the node
  // If the neighbor hasn't been visited yet
  // Add it to the block queue
  // If full, add it to the global queue

  // Calculate space for block queue to go into global queue

  // Store block queue in global queue
}

__global__ void gpu_warp_queuing_kernel(unsigned int *nodePtrs,
                                        unsigned int *nodeNeighbors,
                                        unsigned int *nodeVisited,
                                        unsigned int *currLevelNodes,
                                        unsigned int *nextLevelNodes,
                                        unsigned int *numCurrLevelNodes,
                                        unsigned int *numNextLevelNodes) {

  // INSERT KERNEL CODE HERE

  // This version uses one queue per warp

  // Initialize shared memory queue

  // Loop over all nodes in the curent level
  // Loop over all neighbors of the node
  // If the neighbor hasn't been visited yet
  // Add it to the warp queue
  // If full, add it to the block queue
  // If full, add it to the global queue

  // Calculate space for warp queue to go into block queue

  // Store warp queue in block queue
  // If full, add it to the global queue

  // Calculate space for block queue to go into global queue
  // Saturate block queue counter
  // Calculate space for global queue

  // Store block queue in global queue
}

/******************************************************************************
 Functions
*******************************************************************************/
// DON NOT MODIFY THESE FUNCTIONS!

void gpu_global_queuing(unsigned int *nodePtrs, unsigned int *nodeNeighbors,
                        unsigned int *nodeVisited, unsigned int *currLevelNodes,
                        unsigned int *nextLevelNodes,
                        unsigned int *numCurrLevelNodes,
                        unsigned int *numNextLevelNodes) {

  const unsigned int numBlocks = 45;
  gpu_global_queuing_kernel << <numBlocks, BLOCK_SIZE>>>
      (nodePtrs, nodeNeighbors, nodeVisited, currLevelNodes, nextLevelNodes,
       numCurrLevelNodes, numNextLevelNodes);
}

void gpu_block_queuing(unsigned int *nodePtrs, unsigned int *nodeNeighbors,
                       unsigned int *nodeVisited, unsigned int *currLevelNodes,
                       unsigned int *nextLevelNodes,
                       unsigned int *numCurrLevelNodes,
                       unsigned int *numNextLevelNodes) {

  const unsigned int numBlocks = 45;
  gpu_block_queuing_kernel << <numBlocks, BLOCK_SIZE>>>
      (nodePtrs, nodeNeighbors, nodeVisited, currLevelNodes, nextLevelNodes,
       numCurrLevelNodes, numNextLevelNodes);
}

void gpu_warp_queuing(unsigned int *nodePtrs, unsigned int *nodeNeighbors,
                      unsigned int *nodeVisited, unsigned int *currLevelNodes,
                      unsigned int *nextLevelNodes,
                      unsigned int *numCurrLevelNodes,
                      unsigned int *numNextLevelNodes) {

  const unsigned int numBlocks = 45;
  gpu_warp_queuing_kernel << <numBlocks, BLOCK_SIZE>>>
      (nodePtrs, nodeNeighbors, nodeVisited, currLevelNodes, nextLevelNodes,
       numCurrLevelNodes, numNextLevelNodes);
}
