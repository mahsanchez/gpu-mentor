#include "hip/hip_runtime.h"
#include <cstdio>
#include <cstdlib>
#include <stdio.h>

#include "template.hu"

#define BLOCK_SIZE 512
#define TILE_SIZE 512

// Ceiling funciton for X / Y.
__host__ __device__ static inline int ceil_div(int x, int y) {
    return (x - 1) / y + 1;
}
/******************************************************************************
 GPU kernels
*******************************************************************************/

/*
 * Sequential merge implementation is given. You can use it in your kernels.
 */
__device__ void merge_sequential(float* A, int A_len, float* B, int B_len, float* C) {
    int i = 0, j = 0, k = 0;

    while ((i < A_len) && (j < B_len)) {
        C[k++] = A[i] <= B[j] ? A[i++] : B[j++];
    }

    if (i == A_len) {
        while (j < B_len) {
            C[k++] = B[j++];
        }
    } else {
        while (i < A_len) {
            C[k++] = A[i++];
        }
    }
}

/*
 * Basic parallel merge kernel using co-rank function
 * A, A_len - input array A and its length
 * B, B_len - input array B and its length
 * C - output array holding the merged elements.
 *      Length of C is A_len + B_len (size pre-allocated for you)
 */
__global__ void gpu_merge_basic_kernel(float* A, int A_len, float* B, int B_len, float* C) {
    /* Your code here */
}

/*
 * Arguments are the same as gpu_merge_basic_kernel.
 * In this kernel, use shared memory to increase the reuse.
 */
__global__ void gpu_merge_tiled_kernel(float* A, int A_len, float* B, int B_len, float* C) {
    /* Your code here */
}

/*
 * gpu_merge_circular_buffer_kernel is optional.
 * The implementation will be similar to tiled merge kernel.
 * You'll have to modify co-rank function and sequential_merge
 * to accommodate circular buffer.
 */
__global__ void gpu_merge_circular_buffer_kernel(float* A, int A_len, float* B, int B_len, float* C) {
    /* Your code here */
}

/******************************************************************************
 Functions
*******************************************************************************/

void gpu_basic_merge(float* A, int A_len, float* B, int B_len, float* C) {
    const int numBlocks = 128;
    gpu_merge_basic_kernel<<<numBlocks, BLOCK_SIZE>>>(A, A_len, B, B_len, C);
}

void gpu_tiled_merge(float* A, int A_len, float* B, int B_len, float* C) {
    const int numBlocks = 128;
    gpu_merge_tiled_kernel<<<numBlocks, BLOCK_SIZE>>>(A, A_len, B, B_len, C);
}

void gpu_circular_buffer_merge(float* A, int A_len, float* B, int B_len, float* C) {
    const int numBlocks = 128;
    gpu_merge_circular_buffer_kernel<<<numBlocks, BLOCK_SIZE>>>(A, A_len, B, B_len, C);
}
